#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>


__global__ void colorToGrayscaleKernel(unsigned char* colorImg, unsigned char* grayImg, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int grayOffset = y * width + x;
        int colorOffset = grayOffset * 3;

        unsigned char r = colorImg[colorOffset + 2];
        unsigned char g = colorImg[colorOffset + 1];
        unsigned char b = colorImg[colorOffset];

        grayImg[grayOffset] = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
    }
}

void colorToGrayscale(cv::Mat& colorImg, cv::Mat& grayImg) {
    unsigned char *d_colorImg, *d_grayImg;

    int sizeColor = colorImg.step * colorImg.rows;
    int sizeGray = grayImg.step * grayImg.rows;

    hipMalloc<unsigned char>(&d_colorImg, sizeColor);
    hipMalloc<unsigned char>(&d_grayImg, sizeGray);

    hipMemcpy(d_colorImg, colorImg.ptr(), sizeColor, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((colorImg.cols + blockSize.x - 1) / blockSize.x,
                  (colorImg.rows + blockSize.y - 1) / blockSize.y);

    colorToGrayscaleKernel<<<gridSize, blockSize>>>(d_colorImg, d_grayImg, colorImg.cols, colorImg.rows);

    hipMemcpy(grayImg.ptr(), d_grayImg, sizeGray, hipMemcpyDeviceToHost);

    hipFree(d_colorImg);
    hipFree(d_grayImg);
}
